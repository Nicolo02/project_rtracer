#include "hip/hip_runtime.h"
#include "render.h"

__global__ void kernelrender(point3_t *device_buffer, int* device_num_samples, int *device_image_width, int *device_image_height, point3_t *device_loc00, point3_t *device_camera_center,
                                       point3_t *device_pixel_delta_u, point3_t *device_pixel_delta_v, sphere_t *device_world)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= *device_image_width || j >= *device_image_height)
    {
        return;
    }

    point3_t pixel_color;
      pixel_color.x=0;
      pixel_color.y=0;
      pixel_color.z=0;

      for (int k = 0; k < *device_num_samples; k++)
      {
        ray_t r = get_ray_sample(i, j, *device_loc00, *device_camera_center, *device_pixel_delta_u, *device_pixel_delta_v);
        pixel_color = vec3_sum(ray_color(r, device_world), pixel_color);
      }


    device_buffer[j * *device_image_width + i] = vec3_div_sc(pixel_color, *device_num_samples);

}

void checkCudaError(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

extern "C" void render(point3_t *host_buffer, int n_samples, int image_width, int image_height, point3_t loc00, point3_t camera_center,
            point3_t pixel_delta_u, point3_t pixel_delta_v, sphere_t *world)
{
    point3_t *device_buffer;
    checkCudaError(hipMalloc((void **)&device_buffer, image_width * image_height * sizeof(point3_t)), "Failed to allocate device_buffer");

    int *device_num_samples;
    checkCudaError(hipMalloc((void **)&device_num_samples, sizeof(int)), "Failed to allocate device_num_samples");
    hipMemcpy(device_num_samples, &n_samples, sizeof(int), hipMemcpyHostToDevice);

    int *device_image_width;
    checkCudaError(hipMalloc((void **)&device_image_width, sizeof(int)), "Failed to allocate device_image_width");
    hipMemcpy(device_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);

    int *device_image_height;
    checkCudaError(hipMalloc((void **)&device_image_height, sizeof(int)), "Failed to allocate device_image_height");
    hipMemcpy(device_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);

    point3_t *device_loc00;
    checkCudaError(hipMalloc((void **)&device_loc00, sizeof(point3_t)), "Failed to allocate device_loc00");
    hipMemcpy(device_loc00, &loc00, sizeof(point3_t), hipMemcpyHostToDevice);

    point3_t *device_camera_center;
    checkCudaError(hipMalloc((void **)&device_camera_center, sizeof(point3_t)), "Failed to allocate device_camera_center");
    hipMemcpy(device_camera_center, &camera_center, sizeof(point3_t), hipMemcpyHostToDevice);

    point3_t *device_pixel_delta_u;
    checkCudaError(hipMalloc((void **)&device_pixel_delta_u, sizeof(point3_t)), "Failed to allocate device_pixel_delta_u");
    hipMemcpy(device_pixel_delta_u, &pixel_delta_u, sizeof(point3_t), hipMemcpyHostToDevice);

    point3_t *device_pixel_delta_v;
    checkCudaError(hipMalloc((void **)&device_pixel_delta_v, sizeof(point3_t)), "Failed to allocate device_pixel_delta_v");
    hipMemcpy(device_pixel_delta_v, &pixel_delta_v, sizeof(point3_t), hipMemcpyHostToDevice);

    sphere_t *device_world;
    checkCudaError(hipMalloc((void **)&device_world, 4 * sizeof(sphere_t)), "Failed to allocate device_world");
    hipMemcpy(device_world, world, sizeof(sphere_t), hipMemcpyHostToDevice);

    kernelrender<<<8, 8>>>(device_buffer, device_num_samples, device_image_width, device_image_height, device_loc00, device_camera_center, device_pixel_delta_u, device_pixel_delta_v, device_world);
    hipDeviceSynchronize();

    hipMemcpy(host_buffer, device_buffer, image_width * image_height * sizeof(point3_t), hipMemcpyDeviceToHost);

    hipFree(device_buffer);
    hipFree(device_num_samples);
    hipFree(device_image_width);
    hipFree(device_image_height);
    hipFree(device_loc00);
    hipFree(device_camera_center);
    hipFree(device_pixel_delta_u);
    hipFree(device_pixel_delta_v);
    hipFree(device_world);

    return;
}